#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "hipblas.h"
#include "generateRayleighScalarNoise.c"


int main() {
    int M,N;
    int kboucle;


    FILE *fichier1;
    fichier1=fopen("../results/results_cuda_gemm_fp64.dat","w");


    /* Debut grand boucle */
    for (kboucle=1 ; kboucle<31 ;kboucle++) {

        M=100*kboucle;
        N=M;

        printf(">>>>> Matrix size %dx%d  <<<<<<\n",M,N);

        int iboucle,jboucle,dim;
        int param=20;
        float *mat_real,*mat_imag;
        float charge;

        hipblasStatus_t status ;

        // Chronometre

        struct timespec tpdeb,tpfin,tpcour;
        clockid_t clock_id=CLOCK_REALTIME;
        int status2;

        float dureeloc,dureetot;
        dureetot=0.0;

        // BLAS

        hipblasOperation_t transa,transb;


        /* CUBLAS */

        float time1,time2;
        hipEvent_t start1,start2, stop;

        hipDoubleComplex cualpha,cubeta;
        hipDoubleComplex *h_A,*h_B;
        hipDoubleComplex *h_C;
        hipDoubleComplex* d_A;
        hipDoubleComplex* d_B;
        hipDoubleComplex* d_C;

        transa=HIPBLAS_OP_N;
        transb=HIPBLAS_OP_N;


        cualpha.x=1.0;
        cualpha.y=0.0;
        cubeta.x=0.0;
        cubeta.y=0.0;

        dim=M;

        mat_real=(float*)calloc(M*N,sizeof(float));
        mat_imag=(float*)calloc(M*N,sizeof(float));

        h_A=(hipDoubleComplex*)calloc(M*N,sizeof(hipDoubleComplex));
        h_B=(hipDoubleComplex*)calloc(M*N,sizeof(hipDoubleComplex));
        h_C=(hipDoubleComplex*)calloc(M*N,sizeof(hipDoubleComplex));

        /* Initialize CUBLAS */

        hipblasHandle_t handle;
        status=hipblasCreate(&handle);

        hipMalloc((void**)&d_A, M*N*sizeof(hipDoubleComplex));
        hipMalloc((void**)&d_B, M*N*sizeof(hipDoubleComplex));
        hipMalloc((void**)&d_C, M*N*sizeof(hipDoubleComplex));
        //status = cublasAlloc(M*N, sizeof(hipDoubleComplex), (void**)&d_A);
        //status = cublasAlloc(M*N, sizeof(hipDoubleComplex), (void**)&d_B);
        //status = cublasAlloc(M*N, sizeof(hipDoubleComplex), (void**)&d_C);
        //  if (status != HIPBLAS_STATUS_SUCCESS) {
        //   fprintf (stderr, "!!!! device memory allocation error (B)\n");
        // return EXIT_FAILURE;
        //}

        printf("Generation de la matrice \n");

        for (iboucle=0 ; iboucle<M*N ; iboucle++) {
            generateRayleighScalarNoise(param,mat_real+iboucle,mat_imag+iboucle);
        }

        printf("Hermitien de la matrice\n");


        for(iboucle=0 ; iboucle<N*N ; iboucle++) {
            h_A[iboucle].x=mat_real[iboucle];
            h_A[iboucle].y=mat_imag[iboucle];
            h_B[iboucle].x=mat_imag[iboucle];
            h_B[iboucle].y=mat_real[iboucle];
        }

        /* %%%%% Calcul avec CULA  %%%%%%%%%*/

        printf("Benchmarking.....\n");

        // Remise a zero et deuxieme chorno

        struct timespec tpdeb2,tpfin2;
        float dureeloc2,dureetot2;

        dureeloc=0.0;
        dureetot=0.0;

        hipEventCreate(&start1);
        hipEventCreate(&stop);
        status2=clock_gettime(clock_id, &tpdeb);

        hipEventCreate(&start1);
        hipEventCreate(&stop);

        // Copie de la matrice dans le GPU

        status = hipblasSetVector(M*N, sizeof(hipDoubleComplex), h_A, 1, d_A, 1);
        status2 = clock_gettime(clock_id, &tpdeb2);

        hipDeviceSynchronize();
        hipEventRecord(start1, 0);

        hipblasZgemm(handle,transa,transb,dim,dim,dim,&cualpha,d_A,dim,d_B,dim,&cubeta,d_C,dim);


        hipDeviceSynchronize();
        hipEventRecord( stop, 0);

        status2=clock_gettime(clock_id, &tpfin2);

        // Copie matrice(vecteurs propres) + valeurs propres dans le host

        status = hipblasGetVector(M*N, sizeof(hipDoubleComplex), d_C, 1, h_C, 1);

        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf (stderr, "!!!! device access error (read C)\n");
            return EXIT_FAILURE;
        }

        status2=clock_gettime(clock_id, &tpfin);
        dureeloc=(float)(tpfin.tv_sec-tpdeb.tv_sec)+(float)(tpfin.tv_nsec-tpdeb.tv_nsec)*1.e-9;
        dureetot=dureetot+dureeloc;

        hipEventElapsedTime( &time1, start1, stop );
        hipEventDestroy( start1 );
        hipEventDestroy( stop );
        printf("Timeur CUDA = %f ms\n",time1);


        printf("Temps CUDA avec transfert = %f ms\n",dureeloc*1000);

        //fprintf(fichier2,"%20.15f\n",time1);

        dureeloc2=(float)(tpfin2.tv_sec-tpdeb2.tv_sec)+(float)(tpfin2.tv_nsec-tpdeb2.tv_nsec)*1.e-9;
        dureetot2=dureetot2+dureeloc2;

        printf("Temps CUDA sans transfert = %f ms\n",dureeloc2*1000);
        // fprintf(fichier3,"%20.15f\n",dureeloc2);

        charge=(float)M;
        charge=8*charge*charge*charge;
        charge=charge/dureeloc2;
        printf("compute power = %f GFLOP\n",charge/1e9);
        fprintf(fichier1,"%20.15f\n",charge/1e9);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        hipblasDestroy ( handle ) ;



        free(mat_real);
        free(mat_imag);

        free(h_A);
        free(h_B);
        free(h_C);

        printf("Fin calcul de la matrice %dx%d\n",M,N);
    }

    fclose(fichier1);

    return 0;
}
